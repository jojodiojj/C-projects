#include "hip/hip_runtime.h"
/* File:     lcs.cu
 * Purpose:  Use CUDA to find the longest common subsequence of two 
 *           input sequences of positive ints.  This version proceeds 
 *           down "anti-diagonals".  It stores all of subsequences:  not 
 *           just the subsequences from the previous two diagonals.
 *
 * Compile:  nvcc -g -G -arch=sm_21 -o lcs lcs.cu 
 * Run:      ./lcs <blocks> <threads_per_block>
 *
 * Input:    number of elements in the first sequence (m)
 *           elements of the first sequence
 *           number of elements in the second sequence (n)
 *           elements of the second sequence
 *
 * Output:   The length of the longest common subsequence and
 *           the actual sequence.
 *
 * Name:     Joseph Tanigawa
 *
 * Date:     4/14/2014
 */

#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

void Usage(char progname[]);
void Handle_args(int argc, char* argv[], int* blocks, int* threads_per_block,
                 int* suppress);
void Get_sequences(int* m, int* n, int** seq1_h, int** seq2_h);        
void Print_results(int suppress, int* big_L, int m, int n, int lcs_max,
                   double elapsed);         
int  Get_seq_len(char prompt[]);
void Read_seq(char prompt[], int seq[], int m);
void Print_seq(char title[], int seq[], int m);

/*-------------------------------------------------------------------*/
/* Kernel to find L entry */
__global__ void Find_L_entry(int L_d[], int seq1_d[], int m, int seq2_d[],
                             int n, int max_diag_len, int diag) {
                             
    int j = threadIdx.x + blockIdx.x*blockDim.x;
    int i = diag - j;    
    int lim1j, lijm1;
    int max_seq = max_diag_len + 1;
    int* L = &L_d[(n+1)*max_seq + max_seq];
    
    if (i >= 0 && i < n)
        if (seq1_d[i] == seq2_d[j]) {
            /* Get previous seq length and increment by 1 */
            L[i*(n+1)*max_seq+j*max_seq] =
            L[(i-1)*(n+1)*max_seq+(j-1)*max_seq] + 1; 
            /* Copy prev seq into cur seq */
            int elem;
            for (elem = 0; elem < L[(i-1)*(n+1)*max_seq+(j-1)*max_seq];
                 elem++) {
                L[i*(n+1)*max_seq+j*max_seq + 1+elem] =
                L[(i-1)*(n+1)*max_seq+(j-1)*max_seq + 1+elem];  
            }      
            /* Add seq[i] to the new seq */
            L[i*(n+1)*max_seq+j*max_seq + 1+elem] = seq1_d[i];            
        } else {
            /* Get length of longest existing seq */
            if (i == 0) lim1j = 0;
            else lim1j = L[(i-1)*(n+1)*max_seq+j*max_seq];
            if (j == 0) lijm1 = 0;
            else lijm1 = L[i*(n+1)*max_seq+(j-1)*max_seq];
            
            if (lim1j >= lijm1) {
                L[i*(n+1)*max_seq+j*max_seq] = lim1j;
                if (lim1j != 0) /* Copy prev seq into cur seq  */
                    for (int elem = 0; elem < lim1j; elem++)
                         L[i*(n+1)*max_seq+j*max_seq + 1+elem] =
                         L[(i-1)*(n+1)*max_seq+j*max_seq + 1+elem];    
            } else {
                L[i*(n+1)*max_seq+j*max_seq] = lijm1;
                if (lijm1 != 0) /* Copy prev seq into cur seq  */
                    for (int elem = 0; elem < lijm1; elem++)
                         L[i*(n+1)*max_seq+j*max_seq + 1+elem] =
                         L[i*(n+1)*max_seq+(j-1)*max_seq + 1+elem];  
            }
        }      
}  /* Find_L_entry */


/* Host code */
int main(int argc, char* argv[]) {
    int blocks, threads_per_block, suppress, m, n, lcs_max;
    int diag_count, max_diag_len, diag, big_L_size;
    double start, finish;  
    
    /* host pointers */
    int* seq1_h;
    int* seq2_h;
    int* big_L_h;
    
    /* device pointers */
    int* seq1_d;
    int* seq2_d;
    int* big_L_d;

    Handle_args(argc, argv, &blocks, &threads_per_block, &suppress);
    Get_sequences(&m, &n, &seq1_h, &seq2_h);   
    
    if (m < n)
       max_diag_len = lcs_max = m;
    else
       max_diag_len = lcs_max = n;
       
    diag_count = m + n - 1;
    big_L_size = (m+1)*(n+1)*(lcs_max+1);
    big_L_h = (int*) calloc(big_L_size, sizeof(int));
    
    /* Allocate seq1, seq2 & big_L in device memory */
    hipMalloc(&seq1_d, m*sizeof(int));
    hipMalloc(&seq2_d, n*sizeof(int));
    hipMalloc(&big_L_d, big_L_size*sizeof(int));  
        
    /* Copy vectors from host memory to device memory */
    hipMemcpy(seq1_d, seq1_h, m*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(seq2_d, seq2_h, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(big_L_d, big_L_h, big_L_size*sizeof(int),
               hipMemcpyHostToDevice);
    
    GET_TIME(start);
    for (diag = 0; diag < diag_count; diag++) {
        Find_L_entry<<<blocks, threads_per_block>>>(big_L_d, seq1_d, m, seq2_d,
                                                    n, max_diag_len, diag);
        hipDeviceSynchronize();
    }
    GET_TIME(finish);
    
    hipMemcpy(big_L_h, big_L_d, big_L_size*sizeof(int),
               hipMemcpyDeviceToHost);
    
    Print_results(suppress, big_L_h, m, n, lcs_max, finish-start);
    
    /* Free device memory */
    hipFree(seq1_d);
    hipFree(seq2_d);
    hipFree(big_L_d);
    
    /* Free host memory */
    free(seq1_h);
    free(seq2_h);
    free(big_L_h);
    return 0;
}   /* main */


/*-----------------------------------------------------------------------------
 * Function:  Usage
 * Purpose:   Print the command line for starting the program and exit
 * In arg:    Name of executable
 */
void Usage(char progname[]) {
   fprintf(stderr, "usage: %s <blocks> <threads_per_block>", progname);
   fprintf(stderr, " <'n'(optional)>\n");
   fprintf(stderr, "The last argument supresses output of the");
   fprintf(stderr, " L data structure\n");
   exit(0);
}  /* Usage */


/*-----------------------------------------------------------------------------
 * Function:     Handle_args
 * Purpose:      Read command line arguments
 * In args:      argc, argv
 * In/out args:  blocks, threads_per_block, suppress
 */
void Handle_args(int argc, char* argv[], int* blocks, int* threads_per_block,
                 int* suppress) {
                  
    if (argc < 3) Usage(argv[0]);
    *blocks = strtol(argv[1], NULL, 10);
    *threads_per_block = strtol(argv[2], NULL, 10);
    
    if (argc == 4 && *argv[3] == 'n')
        *suppress = 1;
    else if (argc == 4 && *argv[3] != 'n')
        Usage(argv[0]);    
    else
        *suppress = 0;
}


/*-----------------------------------------------------------------------------
 * Function:     Get_sequences
 * Purpose:      Read in size of sequences and the sequences through user input
 * In args:      argc, argv
 * In/out args:  blocks, threads_per_block, suppress
 */
 void Get_sequences(int* m, int* n, int** seq1_h, int** seq2_h) {
 
    *m = Get_seq_len("first sequence");
    *seq1_h = (int*) malloc(*m*sizeof(int));
    Read_seq("first sequence", *seq1_h, *m);
    Print_seq("1", *seq1_h, *m);
    *n = Get_seq_len("second sequence");
    *seq2_h = (int*) malloc(*n*sizeof(int));
    Read_seq("second sequence", *seq2_h, *n);
    Print_seq("2", *seq2_h, *n);
    printf("\n");
} 


/*-----------------------------------------------------------------------------
 * Function:     Print_results
 * Purpose:      Print results to screen including L if necessary
 * In args:      suppress, big_L, m, n, lcs_max, elapsed
 */
void Print_results(int suppress, int* big_L, int m, int n, int lcs_max,
                   double elapsed) {
                   
    int i, j, L_size, p;
    int* L;
    int* lcs;
    
    L_size = m*n*(lcs_max+1);
    L = &big_L[1*(n+1)*(lcs_max+1) + 1*(lcs_max+1)];
    if (suppress == 0) {
        printf("L =\n");
        for (i = 0; i < L_size+(m-1)*(lcs_max+1); i+=(lcs_max+1)) {
            for (j = i; j < i+lcs_max+1; j++)
                printf("%d ", L[j]);
            printf("\n");
        }    
        printf("\n");
    }
    
    p = L[(m-1)*(n+1)*(lcs_max+1) + (n-1)*(lcs_max+1)];
    lcs = &L[(m-1)*(n+1)*(lcs_max+1) + (n-1)*(lcs_max+1) + 1];       
    printf("The longest common subsequence contains %d elements\n", p);
    Print_seq("longest common", lcs, p);
    printf("Elapsed time for determining LCS = %e seconds\n", elapsed);                   
}


/*-----------------------------------------------------------------------------
 * Function:  Get_seq_len
 * Purpose:   Prompt for the length of a sequence, read it from stdin 
 *            and return it
 * In arg:    prompt:  the name of the sequence
 * Ret val:   the length of the sequence
 */
int Get_seq_len(char prompt[])  {
    int len;

    printf("How many elements in %s?\n", prompt);
    scanf("%d", &len);
    return len;
}   /* Get_seq_len */


/*-------------------------------------------------------------------
 * Function:  Read_seq
 * Purpose:   Read the elements of a sequence from stdin
 * In args:   prompt:  name of the sequence
 *            m:  number of elements in the sequence
 * Out arg:   seq:  the elements
 */
void Read_seq(char prompt[], int seq[], int m) {
   int i;
   printf("Enter the elements of %s\n", prompt);
   for (i = 0; i < m; i++)
      scanf("%d", &seq[i]);
}  /* Read_seq */


/*-------------------------------------------------------------------
 * Function:  Print_seq
 * Purpose:   Print the elements of a sequence to stdout
 * In args:   title:  name of the sequence
 *            seq:  the sequence
 *            m:  the number of elements in the sequence
 */
void Print_seq(char title[], int seq[], int m) {
   int i;
   printf("Sequence %s:\n", title);
   for (i = 0; i < m; i++)
      printf("%d ", seq[i]);
   printf("\n");
}  /* Print_seq */
